#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <memory>

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

texture<uchar4, 2, hipReadModeElementType> tex;

void read_file(std::unique_ptr<uchar4[]>& image,
               int32_t& w,
               int32_t& h,
               const std::string& in_file) {
    std::ifstream input_file(in_file, std::ios::in | std::ios::binary);

    input_file.read(reinterpret_cast<char*>(&w), sizeof(int32_t));
    input_file.read(reinterpret_cast<char*>(&h), sizeof(int32_t));

    image = std::unique_ptr<uchar4[]>(new uchar4[w * h]);

    input_file.read(reinterpret_cast<char*>(image.get()), sizeof(uchar4) * w * h);

    input_file.close();
}

void write_file(const std::unique_ptr<uchar4[]>& image,
                const int32_t& w,
                const int32_t& h,
                const std::string& out_file) {
    std::ofstream output_file(out_file, std::ios::out | std::ios::binary);

    output_file.write(reinterpret_cast<const char*>(&w), sizeof(int32_t));
    output_file.write(reinterpret_cast<const char*>(&h), sizeof(int32_t));
    output_file.write(reinterpret_cast<const char*>(image.get()), sizeof(uchar4) * w * h);

    output_file.close();
}

__device__
double convert_greyscale(uchar4 pixel) {
    // Alpha ignored
	return 0.299 * pixel.x + 
           0.587 * pixel.y + 
           0.114 * pixel.z;
}

__global__
void kernel(uchar4 *image, int32_t w, int32_t h) {
    int idx     = blockDim.x * blockIdx.x + threadIdx.x;
	int idy     = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;

    int x_filter[3][3] = {
        {-1,  0,  1},
        {-1,  0,  1},
        {-1,  0,  1}
    };
    int y_filter[3][3] = {
        {-1, -1, -1},
        { 0,  0,  0},
        { 1,  1,  1}
    };

	for(int y = idy; y < h; y += offsety) {
		for(int x = idx; x < w; x += offsetx) {
            double gx = 0;
            double gy = 0;

            for (int i = 0; i < 3; ++i) {
                for (int j = 0; j < 3; ++j) {
                    double grey_pixel = convert_greyscale( tex2D(tex, x + i - 1, y + j - 1) );
                    gx += x_filter[i][j] * grey_pixel;
                    gy += y_filter[i][j] * grey_pixel;
                }
            }

            // Must be 0 to 255 inclusive, else overflow
            unsigned char g = min( sqrt(gx * gx + gy * gy), 255.0 );

            int offset = y * w + x;
            image[offset].x = g;
            image[offset].y = g;
            image[offset].z = g;
            image[offset].w = 0;
		}
    }

}

int main() {
    std::string in_file, out_file;
    std::cin >> in_file >> out_file;

    int32_t w, h;
    // ama big boy
    std::unique_ptr<uchar4[]> image;

    read_file(image, w, h, in_file);

    hipArray *arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&arr, &ch, w, h));
	CSC(hipMemcpyToArray(arr, 0, 0, image.get(), sizeof(uchar4) * w * h, hipMemcpyHostToDevice));

    tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.channelDesc = ch;
	tex.filterMode = hipFilterModePoint;
	tex.normalized = false;
	CSC(hipBindTextureToArray(tex, arr, ch));

	uchar4 *dev_image;
	CSC(hipMalloc(&dev_image, sizeof(uchar4) * w * h));

    kernel<<< dim3(32, 32), dim3(32, 32) >>>(dev_image, w, h);

	CSC(hipMemcpy(image.get(), dev_image, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));

	CSC(hipUnbindTexture(tex));

	CSC(hipFreeArray(arr));
	CSC(hipFree(dev_image));

    write_file(image, w, h, out_file);
}
