#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <memory>
#include <math.h>
#include <random>

#include <GL/glew.h>
#include <GL/freeglut.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#define ESC 27
#define SPACEBAR 32

#define sqr3(x) ((x)*(x)*(x))
#define sqr(x)  ((x)*(x))

struct Particle {
    float x;
    float y;
    float z;

    float dx;
    float dy;
    float dz;

    float q;
};

struct Player {
    Player() {
        x = -1.5;
        y = -1.5;
        z = 1.0;
    }

    float x;
    float y;
    float z;

    float dx;
    float dy;
    float dz;

    float yaw;
    float pitch;

    float dyaw;
    float dpitch;

    const float top_speed = 0.3;
};



namespace {
    int w = 1024;
    int h = 648;

    bool keystates[256] = {};

    const unsigned int particle_count = 50;
    const unsigned int floor_percision = 100;
    const float half_len = 15.0; // Half the length of cube edge

    hipGraphicsResource *res;
    GLuint floor_texture;
    GLuint quad_texture;
    GLuint vbo;

    GLUquadric *quadratic;

    std::vector<Particle> particles;
    Particle *d_particles;

    Particle cam_particle;

    Player player;
}



__global__ 
void recalc_particle_velocity(Particle *particles, unsigned int count, 
            Particle cam_part, Particle player_part,
            float w, float e0, float dt, float k, float grav) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;

    for (unsigned int p = idx; p < count; p += offsetx) {
        Particle& part = particles[p];

        part.dx *= w;
        part.dy *= w;
        part.dz *= w;

        float dx_sum = 0;
        float dy_sum = 0;
        float dz_sum = 0;
        float l;
        float coef;

        // Other particles
        for (unsigned int p_other = 0; p_other < count; ++p_other) {
            if (p_other == p) {
                continue;
            }
            Particle &other = particles[p_other];

            l = sqrt(sqr(part.x - other.x) + sqr(part.y - other.y) + sqr(part.z - other.z));
            coef = other.q / (l * l * l + e0);
            dx_sum += coef * (part.x - other.x);
            dy_sum += coef * (part.y - other.y);
            dz_sum += coef * (part.z - other.z);
        }

        // Walls
        dx_sum += part.q * (part.x - half_len) / (sqr3(fabs(part.x - half_len)) + e0);
        dx_sum += part.q * (part.x + half_len) / (sqr3(fabs(part.x + half_len)) + e0);

        dy_sum += part.q * (part.y - half_len) / (sqr3(fabs(part.y - half_len)) + e0);
        dy_sum += part.q * (part.y + half_len) / (sqr3(fabs(part.y + half_len)) + e0);

        dz_sum += part.q * (part.z - 2 * half_len) / (sqr3(fabs(part.z - 2 * half_len)) + e0);
        dz_sum += part.q * (part.z + 0.0) / (sqr3(fabs(part.z + 0.0)) + e0);

        // Cam
        l = sqrt(sqr(part.x - cam_part.x) + sqr(part.y - cam_part.y) + sqr(part.z - cam_part.z));
        coef = cam_part.q / (l * l * l + e0);
        dx_sum += coef * (part.x - cam_part.x);
        dy_sum += coef * (part.y - cam_part.y);
        dz_sum += coef * (part.z - cam_part.z);

        // Player
        l = sqrt(sqr(part.x - player_part.x) + sqr(part.y - player_part.y) + sqr(part.z - player_part.z));
        coef = player_part.q / (l * l * l + e0);
        dx_sum += coef * (part.x - player_part.x);
        dy_sum += coef * (part.y - player_part.y);
        dz_sum += coef * (part.z - player_part.z);

        dx_sum *= part.q * k * dt;
        dy_sum *= part.q * k * dt;
        dz_sum *= part.q * k * dt;

        part.dx += dx_sum;
        part.dy += dy_sum;
        part.dz += dz_sum - grav * dt;

        float new_x = part.x + part.dx * dt;
        float new_y = part.y + part.dy * dt;
        float new_z = part.z + part.dz * dt;

        part.x += part.dx * dt * (abs(new_x) < half_len - e0);
        part.y += part.dy * dt * (abs(new_y) < half_len - e0);
        part.z += part.dz * dt * (e0 < new_z && new_z < 2 * half_len - e0);
    }
}

__global__
void calc_floor(uchar4 *data, Particle *particles, unsigned int count,
        Particle cam_part, float e0, float z_shift, float k) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;

    for (int i = idx; i < floor_percision; i += offsetx) {
        for (int j = idy; j < floor_percision; j += offsety) {
            float x = (2.0 * i / (floor_percision - 1.0) - 1.0) * half_len;
            float y = (2.0 * j / (floor_percision - 1.0) - 1.0) * half_len;
            float voltage = 0;
            for (unsigned int p = 0; p < count; ++p) {
                Particle &part = particles[p];
                voltage += part.q / (sqr(part.x - x) + sqr(part.y - y) + sqr(part.z - z_shift) + e0);
            }
            voltage += cam_part.q / (sqr(cam_part.x - x) + sqr(cam_part.y - y) + sqr(cam_part.z - z_shift) + e0);
            voltage *= k;
            data[j * floor_percision + i] = make_uchar4(min((int)voltage, 255), 0, 0, 255);
        }
    }
}

void display() {
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();

    gluPerspective(90.0, (GLfloat)w/(GLfloat)h, 0.1, 100.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    float cos_pitch = cos(player.pitch);
    gluLookAt(player.x, player.y, player.z,
              player.x + cos(player.yaw) * cos_pitch,
              player.y + sin(player.yaw) * cos_pitch,
              player.z + sin(player.pitch),
              0.0f, 0.0f, 1.0f);

    glBindTexture(GL_TEXTURE_2D, quad_texture);
    static float angle = 0.0;
    for (const auto& p : particles) {
        glPushMatrix();
            glTranslatef(p.x, p.y, p.z); 
            glRotatef(angle, 0.0, 0.0, 1.0);
            gluSphere(quadratic, 0.625f, 8, 8);
        glPopMatrix();
    }
    glPushMatrix();
        glTranslatef(cam_particle.x, cam_particle.y, cam_particle.z); 
        glRotatef(angle, 0.0, 0.0, 1.0);
        gluSphere(quadratic, 0.625f, 8, 8);
    glPopMatrix();
    angle += 0.15;

    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, vbo);
    glBindTexture(GL_TEXTURE_2D, floor_texture);
    glTexImage2D(GL_TEXTURE_2D, 0, 3, (GLsizei)floor_percision, (GLsizei)floor_percision, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    glBegin(GL_QUADS);
        glTexCoord2f(0.0, 0.0);
        glVertex3f(-half_len, -half_len, 0.0);

        glTexCoord2f(1.0, 0.0);
        glVertex3f(half_len, -half_len, 0.0);

        glTexCoord2f(1.0, 1.0);
        glVertex3f(half_len, half_len, 0.0);

        glTexCoord2f(0.0, 1.0);
        glVertex3f(-half_len, half_len, 0.0);
    glEnd();

    glBindTexture(GL_TEXTURE_2D, 0);

    glLineWidth(2);
    glColor3f(0.5f, 0.5f, 0.5f);
    glBegin(GL_LINES);
        glVertex3f(-half_len, -half_len, 0.0);
        glVertex3f(-half_len, -half_len, 2.0 * half_len);

        glVertex3f(half_len, -half_len, 0.0);
        glVertex3f(half_len, -half_len, 2.0 * half_len);

        glVertex3f(half_len, half_len, 0.0);
        glVertex3f(half_len, half_len, 2.0 * half_len);

        glVertex3f(-half_len, half_len, 0.0);
        glVertex3f(-half_len, half_len, 2.0 * half_len);
    glEnd();

    glBegin(GL_LINE_LOOP);
        glVertex3f(-half_len, -half_len, 0.0);
        glVertex3f( half_len, -half_len, 0.0);
        glVertex3f( half_len,  half_len, 0.0);
        glVertex3f(-half_len,  half_len, 0.0);
    glEnd();

    glBegin(GL_LINE_LOOP);
        glVertex3f(-half_len, -half_len, 2.0 * half_len);
        glVertex3f( half_len, -half_len, 2.0 * half_len);
        glVertex3f( half_len,  half_len, 2.0 * half_len);
        glVertex3f(-half_len,  half_len, 2.0 * half_len);
    glEnd();

    glColor3f(1.0f, 1.0f, 1.0f);

    glutSwapBuffers();
}

void key_down(unsigned char key, int x, int y) {
    keystates[key] = true;
}

void key_up(unsigned char key, int x, int y) {
    keystates[key] = false;
}

void process_keys() {
    float top_speed = player.top_speed;

    if (keystates['w']) {
        float cos_pitch = cos(player.pitch);
        player.dx += cos(player.yaw) * cos_pitch * top_speed;
        player.dy += sin(player.yaw) * cos_pitch * top_speed;
        player.dz += sin(player.pitch) * top_speed;
    }
    if (keystates['s']) {
        float cos_pitch = cos(player.pitch);
        player.dx -= cos(player.yaw) * cos_pitch * top_speed;
        player.dy -= sin(player.yaw) * cos_pitch * top_speed;
        player.dz -= sin(player.pitch) * top_speed;
    }
    if (keystates['a']) {
        player.dx += -sin(player.yaw) * top_speed;
        player.dy += cos(player.yaw) * top_speed;
    }
    if (keystates['d']) {
        player.dx += sin(player.yaw) * top_speed;
        player.dy += -cos(player.yaw) * top_speed;
    }
    if (keystates[SPACEBAR]) {
        player.dz += top_speed;
    }
    if (keystates['c']) {
        player.dz -= top_speed;
    }
    if (keystates[ESC]) {
        hipGraphicsUnregisterResource(res);
        glDeleteTextures(1, &floor_texture);
        glDeleteTextures(1, &quad_texture);
        glDeleteBuffers(1, &vbo);
        gluDeleteQuadric(quadratic);
        hipFree(d_particles);
        exit(0);
    }
}

void init_cam_particle() {
    cam_particle.x = 0.0f;
    cam_particle.y = 0.0f;
    cam_particle.z = -200.0f;

    cam_particle.dx = 0;
    cam_particle.dy = 0;
    cam_particle.dz = 0;
}

void shoot_cam_particle() {
    cam_particle.x = player.x;
    cam_particle.y = player.y;
    cam_particle.z = player.z;

    float speed = 30.0;
    float cos_pitch = cos(player.pitch);
    cam_particle.dx = speed * cos(player.yaw) * cos_pitch;
    cam_particle.dy = speed * sin(player.yaw) * cos_pitch;
    cam_particle.dz = speed * sin(player.pitch);

    cam_particle.q = 50;
}

void process_cam_particle(float dt) {
    cam_particle.x += cam_particle.dx * dt;
    cam_particle.y += cam_particle.dy * dt;
    cam_particle.z += cam_particle.dz * dt;

    float box_limit = half_len + 100.0;
    if (abs(cam_particle.x) >= box_limit || 
            abs(cam_particle.y) >= box_limit ||
            abs(cam_particle.z) >= half_len + box_limit) {
        init_cam_particle();
    }
}

void update() {
    process_keys();

    const float speed = player.top_speed;
    float v = sqrt(player.dx * player.dx + player.dy * player.dy + player.dz * player.dz);
    if (v > speed) {
        float norm = speed / v;
        player.dx *= norm;
        player.dy *= norm;
        player.dz *= norm;
    }

    float slow_down = 0.99;
    player.x += player.dx;
    player.y += player.dy;
    player.z += player.dz;
    player.dx *= slow_down;
    player.dy *= slow_down;
    player.dz *= slow_down;

    if (player.z < 1.0) {
        player.z = 1.0;
        player.dz = 0.0;
    }

    if (fabs(player.dpitch) + fabs(player.dyaw) > 0.00001) {
        player.yaw += player.dyaw;
        player.pitch += player.dpitch;
        player.pitch = min(M_PI / 2.0 - 0.0001, max(-M_PI / 2.0 + 0.0001, player.pitch));
        player.dyaw = player.dpitch = 0.0;
    }

    float w = 0.999, e0 = 1e-3, dt = 0.01, z_shift = 0.75, k = 50.0, gravity = 10.0;

    process_cam_particle(dt);
    Particle player_particle;
    player_particle.x = player.x;
    player_particle.y = player.y;
    player_particle.z = player.z;
    player_particle.q = 10;

    hipMemcpy(d_particles, particles.data(), sizeof(Particle) * particles.size(), hipMemcpyHostToDevice);

    recalc_particle_velocity<<<256, 256>>>(d_particles, particles.size(), 
            cam_particle, player_particle, w, e0, dt, k, gravity);

    hipMemcpy(particles.data(), d_particles, sizeof(Particle) * particles.size(), hipMemcpyDeviceToHost);

    uchar4 *dev_data;
    size_t size;
    hipGraphicsMapResources(1, &res, 0);
    hipGraphicsResourceGetMappedPointer((void**)&dev_data, &size, res);

    calc_floor<<<dim3(32, 32), dim3(32, 8)>>>(dev_data, d_particles, particles.size(),
            cam_particle, e0, z_shift, k);

    hipGraphicsUnmapResources(1, &res, 0);

    glutPostRedisplay();
}

void mouse(int x, int y) {
    if (x != w/2 || y != h/2) {
        glutWarpPointer(w / 2, h / 2);
        float norm_coef = 0.001;
        float dx = norm_coef * (x - w/2);
        float dy = norm_coef * (y - h/2);
        player.dyaw -= dx;
        player.dpitch -= dy;
    }
}

void mouse_press(int button, int state, int x, int y) {
    if (state != GLUT_DOWN) {
        return;
    }

    if (button == GLUT_LEFT_BUTTON) {
        shoot_cam_particle();
    } else if (button == GLUT_RIGHT_BUTTON) {
        player.dx = 0;
        player.dy = 0;
        player.dz = 0;
    }
}

void reshape(int w_new, int h_new) {
    w = w_new;
    h = h_new;
    glViewport(0, 0, w, h);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
}

std::vector<Particle> 
fill_with_random_particles(unsigned int particle_count) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(-half_len, half_len);

    std::vector<Particle> particles(particle_count);

    for (auto& p : particles) {
        p.x = dist(gen);
        p.y = dist(gen);
        p.z = half_len + dist(gen);
        p.q = 1;
    }

    return particles;
}

void read_file(std::unique_ptr<unsigned char[]>& data,
               int& w,
               int& h,
               const std::string& in_file) {
    std::ifstream input_file(in_file, std::ios::in | std::ios::binary);

    input_file.read(reinterpret_cast<char*>(&w), sizeof(int));
    input_file.read(reinterpret_cast<char*>(&h), sizeof(int));

    data = std::unique_ptr<unsigned char[]>(new unsigned char[4 * w * h]);

    input_file.read(reinterpret_cast<char*>(data.get()), sizeof(unsigned char) * 4 * w * h);

    input_file.close();
}

void setup_glut(int *main_argc, char **main_argv) {
    glutInit(main_argc, main_argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
    glutInitWindowSize(w, h);
    glutCreateWindow("Particle simulator 2021");

    glutIdleFunc(update);
    glutDisplayFunc(display);
    glutKeyboardFunc(key_down);
    glutKeyboardUpFunc(key_up);
    glutMotionFunc(mouse);
    glutPassiveMotionFunc(mouse);
    glutMouseFunc(mouse_press);
    glutReshapeFunc(reshape);

    glutSetCursor(GLUT_CURSOR_NONE);
}

void setup_gl() {
    glEnable(GL_TEXTURE_2D);
    glShadeModel(GL_SMOOTH);
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    glClearDepth(1.0f);
    glDepthFunc(GL_LEQUAL);
    glEnable(GL_DEPTH_TEST);
    glEnable(GL_CULL_FACE);

    std::unique_ptr<unsigned char[]> data;
    int wt, ht;
    read_file(data, wt, ht, "in.data");

    glGenTextures(1, &quad_texture);
    glBindTexture(GL_TEXTURE_2D, quad_texture);
    glTexImage2D(GL_TEXTURE_2D, 0, 3, (GLsizei)wt, (GLsizei)ht, 0, GL_RGBA, GL_UNSIGNED_BYTE, (void*)data.get());
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

    quadratic = gluNewQuadric();
    gluQuadricTexture(quadratic, GL_TRUE);

    glGenTextures(1, &floor_texture);
    glBindTexture(GL_TEXTURE_2D, floor_texture);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    glewInit();
    glGenBuffers(1, &vbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, vbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, 
                 floor_percision * floor_percision * sizeof(uchar4), 
                 NULL,
                 GL_DYNAMIC_DRAW);
    hipGraphicsGLRegisterBuffer(&res, vbo, cudaGraphicsMapFlagsWriteDiscard);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
}

int main(int argc, char *argv[]) {
    setup_glut(&argc, argv);
    setup_gl();

    particles = fill_with_random_particles(particle_count);
    init_cam_particle();
    // Two more for camera shot particle and for player particle
    hipMalloc(&d_particles, sizeof(Particle) * particle_count);

    glutMainLoop();
}
